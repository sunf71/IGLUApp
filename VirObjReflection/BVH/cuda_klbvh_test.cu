#include "hip/hip_runtime.h"
#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include "cuda_klbvh.h"
#include "MortonCode.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
 #include <thrust/count.h>
#include <thrust/execution_policy.h>
#include "cuda_klbvh.cuh"
#include <algorithm>
#include <cstdlib>

#include "gputimer.cuh"
#include "timer.h"
using namespace nih;


texture<uint32> bvhTex;
struct bvhTexHelper
{
	static const uint32 nodeSize = 11;
	static const uint32 LChildOf = 6;
	static const uint32 RChildOf = 7;
	static const uint32 pidOf = 8;
	static const uint32 leafStartOf = 9;
	static const uint32 leafEndOf = 10;
	float p[6];
	NIH_DEVICE float* getBbox(uint32 id)
	{
		uint32 offset = id*nodeSize;
		uint32 t[6];
		
		for(int i=0; i<6; i++)
		{
			t[i] = tex1Dfetch(bvhTex,offset+i);
			p[i] = bitsToFloat(t[i]);
		}

	
		return p;
	}

	NIH_DEVICE uint32 getLChild(uint32 id)
	{
		return tex1Dfetch(bvhTex,id*nodeSize+LChildOf);
	}

	NIH_DEVICE uint32 getRChild(uint32 id)
	{
		return tex1Dfetch(bvhTex,id*nodeSize+RChildOf);
	}
	NIH_DEVICE uint32 getPid(uint32 id)
	{
		return tex1Dfetch(bvhTex,id*nodeSize+pidOf);
	}
	NIH_DEVICE uint32 getleafStart(uint32 id)
	{
		return tex1Dfetch(bvhTex,id*nodeSize+leafStartOf);
	}
	NIH_DEVICE uint32 getleafEnd(uint32 id)
	{
		return tex1Dfetch(bvhTex,id*nodeSize+leafEndOf);
	}
	NIH_DEVICE bool isLeaf(uint32 id)
	{
		return getleafStart(id) == getleafEnd(id);
	}
};




NIH_DEVICE bool AABBOverlap(Bbox3f& boxA, Bbox3f& boxB)
{
	for (int i=0; i<3; i++)
	{
		if (fabs(boxB.m_max[i]+boxB.m_min[i]-boxA.m_max[i]-boxA.m_min[i]) <
			boxA.m_max[i]-boxA.m_min[i] + boxB.m_max[i] - boxB.m_min[i])
			return true;
	}
	return false;
}

FORCE_INLINE NIH_DEVICE void FrustumCulling(TriFrustum& frustum, uint32 frustumId,
	uint32 priSize,
	cullingContext* out)
{
	bvhTexHelper helper;
	uint32 offset = priSize*frustumId;
	const uint32 stack_size  = 64;
	uint32 stack[stack_size];
	uint32 top = 0;
	stack[top++] = 0;
	while(top>0)
	{
		uint32 idx = stack[--top];
		//Bintree_node * node = &bvh[idx];
		uint32 RChild = helper.getRChild(idx);
		uint32 LChild = helper.getLChild(idx);
		
		int ret = Intersect(frustum,helper.getBbox(idx));
		if (ret == 2)
		{
			//�ཻ
			
			if(helper.isLeaf(RChild))
			{
				if (Intersect(frustum,helper.getBbox(RChild)))
				{
					uint32 pid = helper.getPid(RChild);
					out[offset+pid].frustumId = frustumId;
					out[offset+pid].triId = pid;					
				}				
			}
			else
				stack[top++] = RChild;

			if (helper.isLeaf(LChild))
			{
				
				if (Intersect(frustum,helper.getBbox(LChild)))
				{
					uint32 pid = helper.getPid(LChild);
					out[offset+pid].frustumId = frustumId;					
					out[offset+pid].triId = pid;
				}				
			}
			else
				stack[top++] = LChild;
		}
		else if (ret == 1)
		{
			//in
			for(int k= helper.getleafStart(idx); k<=helper.getleafEnd(idx);k++)
			{	
				out[offset+k].frustumId = frustumId;
				out[offset+k].triId = k;
			}

		}
	}
}


__global__ void FrustumCullingKernel(TriFrustum* frustumP, int frustum_num, uint32 priSize,cullingContext* list)
{
	uint32 step = blockDim.x * gridDim.x;
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
		i < frustum_num; 
		i += step) 
	{
		TriFrustum frustum = frustumP[i];
		FrustumCulling(frustum,i,priSize,list);
		//FrustumCullingT(frustum,i,bvh,priSize,list);
	}
}

void NIH_HOST_DEVICE GenerateVirFrustum(uint32 id, const Vector3f& eye,const Vector3f& p1,const Vector3f& p2, const Vector3f& p3, float farD, TriFrustum& frustum)
{
	     //��5��ƽ�淽��
		//��׶ƽ�淨��ָ����׶��
		plane_t pTri(p1,p2,p3);	
		
		float d  = pTri.distance(eye);
		//�ӵ㲻��λ��������ƽ�淨����һ��
		if (d<= 0)
			return;

		//�����ӵ�
		Vector3f fNormal(pTri.a,pTri.b,pTri.c);
		float dir = dot(eye-p1,fNormal);
		Vector3f vEye = eye-fNormal*2.f*dir;
		
		frustum.id = id;
		frustum.planes[0] = plane_t(eye,p2,p1);
		frustum.planes[1] = plane_t(eye,p3,p2);
		frustum.planes[2] = plane_t(eye,p1,p3);
		frustum.planes[3] =  plane_t(p1,p3,p2);
		frustum.planes[4] = pTri;
		Vector3f c = (p1+p2+p3)*1.f/3.f;
		float cosT = (-d)/euclidean_distance(eye,c);
		frustum.planes[4].d -= farD/cosT;		
}

//����׶����kernel
__global__ void GenerateVirFrustumKernel(Vector3f* eye,Vector3f* p123, TriFrustum* frustums,float farD, int count)
{
	uint32 step = blockDim.x * gridDim.x;
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
		i < count; 
		i += step) 
	{
		GenerateVirFrustum(i,*eye,p123[i*3],p123[i*3+1],p123[i*3+2],farD,frustums[i]);
		
	}
}



bool BboxCompare(const Bbox3f& lbox, const Bbox3f& rbox)
{
	const double zero = 0.0001;

	return (abs(lbox.m_min[0]-rbox.m_min[0])<zero &&
		abs(lbox.m_min[1]-rbox.m_min[1])<zero && 
		abs(lbox.m_min[2]-rbox.m_min[2])<zero &&
		abs(lbox.m_max[0]-rbox.m_max[0])<zero &&
		abs(lbox.m_max[1]-rbox.m_max[1])<zero && 
		abs(lbox.m_max[2]-rbox.m_max[2])<zero );
}


