#include "hip/hip_runtime.h"
#include "CudaFunctions.h"
#include <thrust/host_vector.h>
#include <thrust/execution_policy.h>
#include "BVH\cuda_klbvh.h"
#include "BVH\cuda_klbvh.cuh"
#include <iostream>

texture<uint32> bvhTex;
//����BVHʱ������ͼԪ����
texture<uint32> indexTex;
//�ü���������壬ע������������Ҫ������ȫ���ڴ���
texture<uint32> virObjTex;

cullingContext* cullingResult;
//����bvhʱ������ͼԪ���
uint32* gd_indices;
void NIH_HOST_DEVICE GenerateVirFrustum(uint32 id, const Vector3f& eye,const Vector3f& p1,const Vector3f& p2, const Vector3f& p3, float farD, TriFrustum& frustum);

__global__ void GenerateVirFrustumKernel(Vector3f* eye,Vector3f* p123, TriFrustum* frustums, float farD, int count);
namespace cuda
{
	struct bvhTexHelper
	{
		static const uint32 nodeSize = 11;
		static const uint32 LChildOf = 6;
		static const uint32 RChildOf = 7;
		static const uint32 pidOf = 8;
		static const uint32 leafStartOf = 9;
		static const uint32 leafEndOf = 10;
		float p[6];
		NIH_DEVICE float* getBbox(uint32 id)
		{
			uint32 offset = id*nodeSize;
			uint32 t[6];

			for(int i=0; i<6; i++)
			{
				t[i] = tex1Dfetch(bvhTex,offset+i);
				p[i] = bitsToFloat(t[i]);
			}


			return p;
		}

		NIH_DEVICE uint32 getLChild(uint32 id)
		{
			return tex1Dfetch(bvhTex,id*nodeSize+LChildOf);
		}

		NIH_DEVICE uint32 getRChild(uint32 id)
		{
			return tex1Dfetch(bvhTex,id*nodeSize+RChildOf);
		}
		NIH_DEVICE uint32 getPid(uint32 id)
		{
			return tex1Dfetch(bvhTex,id*nodeSize+pidOf);
		}
		NIH_DEVICE uint32 getleafStart(uint32 id)
		{
			return tex1Dfetch(bvhTex,id*nodeSize+leafStartOf);
		}
		NIH_DEVICE uint32 getleafEnd(uint32 id)
		{
			return tex1Dfetch(bvhTex,id*nodeSize+leafEndOf);
		}
		NIH_DEVICE bool isLeaf(uint32 id)
		{
			return getleafStart(id) == getleafEnd(id);
		}
	};

	FORCE_INLINE NIH_DEVICE void FrustumCulling(TriFrustum& frustum, uint32 frustumId,
		uint32 priSize,
		cullingContext* out)
	{
		bvhTexHelper helper;
		uint32 offset = priSize*frustumId;
		const uint32 stack_size  = 64;
		uint32 stack[stack_size];
		uint32 top = 0;
		stack[top++] = 0;
		while(top>0)
		{
			uint32 idx = stack[--top];
			//Bintree_node * node = &bvh[idx];
			uint32 RChild = helper.getRChild(idx);
			uint32 LChild = helper.getLChild(idx);

			int ret = Intersect(frustum,helper.getBbox(idx));
			if (ret == 2)
			{
				//�ཻ

				if(helper.isLeaf(RChild))
				{
					if (Intersect(frustum,helper.getBbox(RChild)))
					{
						uint32 pid = helper.getPid(RChild);
						pid = tex1Dfetch(indexTex,pid);
						out[offset+pid].frustumId = frustumId;
						out[offset+pid].triId = pid;					
					}				
				}
				else
					stack[top++] = RChild;

				if (helper.isLeaf(LChild))
				{

					if (Intersect(frustum,helper.getBbox(LChild)))
					{
						uint32 pid = helper.getPid(LChild);
						pid = tex1Dfetch(indexTex,pid);
						out[offset+pid].frustumId = frustumId;					
						out[offset+pid].triId = pid;
					}				
				}
				else
					stack[top++] = LChild;
			}
			else if (ret == 1)
			{
				//in
				for(int k= helper.getleafStart(idx); k<=helper.getleafEnd(idx);k++)
				{	
					
					uint32	pid = tex1Dfetch(indexTex,k);
					out[offset+pid].frustumId = frustumId;
					out[offset+pid].triId = pid;
				}

			}
		}
	}


	__global__ void FrustumCullingKernel(TriFrustum* frustumP, int frustum_num, uint32 priSize,cullingContext* list)
	{
		uint32 step = blockDim.x * gridDim.x;
		for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
			i < frustum_num; 
			i += step) 
		{
			TriFrustum frustum = frustumP[i];
			FrustumCulling(frustum,i,priSize,list);
			//FrustumCullingT(frustum,i,bvh,priSize,list);
		}
	}
	__device__  __host__ nih::Vector3f MatrixXVector3f(const float* mat, nih::Vector3f& vec)
	{
		float* d= &vec[0];
		float tmp[4];
		tmp[0] = mat[0]*d[0] + mat[4]*d[1] + mat[8]*d[2] + mat[12];
		tmp[1] = mat[1]*d[0] + mat[5]*d[1] + mat[9]*d[2] + mat[13];
		tmp[2] = mat[2]*d[0] + mat[6]*d[1] + mat[10]*d[2] + mat[14];
		tmp[3] = mat[3]*d[0] + mat[7]*d[1] + mat[11]*d[2] + mat[15];

		Vector3f ret( tmp );
		ret /= tmp[3];
		return ret;
	}


	size_t GridSize(size_t jobSize,size_t blockSize = 128)
	{

		int numSMs;
		hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);
		size_t max_blocks = 65535;
		return numSMs*nih::min( max_blocks, (jobSize + (blockSize*numSMs)-1) / (blockSize*numSMs) );
	}

	//@vertices �Ƕ������飬ÿ��������3����������ʾ
	//@indices �������ζ�����������飬ÿ����������3������
	//@matrix ��4��4ģ�;���
	//@points ������������������ζ���
	//@centers ������������������ε�����
	//@boxes ������������������εİ�Χ��
	//@size �����θ���
	__global__ void LoadOBJKernel(float* vertices,
		uint32* indices,
		float* matrix,
		nih::Vector3f* points,
		nih::Vector3f* centers,
		nih::Bbox3f* boxes,
		uint32 size)
	{
		uint32 step = blockDim.x * gridDim.x;
		for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
			i < size; 
			i += step) 
		{
			uint32 tOffset = 3*i;


			for( int j=0; j<3; j++)
			{
				for( int k=0; k<3; k++)
				{
					points[tOffset+j][k] = vertices[indices[tOffset+j]*3+k];
					centers[i][k] += points[tOffset+j][k];
				}
				boxes[i].insert(points[tOffset+j]);				
			}			
			centers[i] /= 3.f;
			centers[i] = MatrixXVector3f(matrix,centers[i]);
			boxes[i].m_min = MatrixXVector3f(matrix,boxes[i].m_min);
			boxes[i].m_max = MatrixXVector3f(matrix,boxes[i].m_max);
		}
	}

	//����ԭ�е������Ͳü���������µ�����
	__global__ void UpdateElementKernel(const unsigned* inPtr, unsigned * outPtr,  cullingContext* cullingResult, int size)
	{
		unsigned step = blockDim.x * gridDim.x;
		for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
			i < size; 
			i += step) 
		{
			
			unsigned offset = i*3;
			unsigned offset2 = tex1Dfetch(virObjTex,i*2+1)*3;
			//unsigned offset2 = cullingResult[i].triId*3;
			outPtr[offset] = inPtr[offset2];
			outPtr[offset+1] = inPtr[offset2+1];
			outPtr[offset+2] = inPtr[offset2+2];
		}
	}
	void LoadOBJReader(iglu::IGLUOBJReader::Ptr obj, 
		iglu::IGLUMatrix4x4::Ptr matrix,
		nih::Vector3f* d_points,
		nih::Vector3f* d_centers,
		nih::Bbox3f* d_boxes)
	{
		size_t size = obj->GetTriangleCount();
		size_t vertSize = obj->GetVaoVerts().size();
		float* vertices = (float*)(&obj->GetVaoVerts()[0]);
		uint32* indices = obj->GetElementArrayData();
		/*for(int i=0; i<size*3; i+=3)
		std::cout<<indices[i]<<","<<indices[i+1]<<","<<indices[i+2]<<std::endl;*/
		float* d_vertices;
		hipMalloc((void**)&d_vertices,sizeof(float)*vertSize);
		hipMemcpy(d_vertices,vertices,sizeof(float)*vertSize,hipMemcpyHostToDevice);

		uint32* d_indices;
		hipMalloc((void**)&d_indices,sizeof(uint32)*3*size);
		hipMemcpy(d_indices,indices,sizeof(uint32)*3*size,hipMemcpyHostToDevice);

		float* d_matrix;
		hipMalloc((void**)&d_matrix,sizeof(float)*16);
		hipMemcpy(d_matrix,matrix->GetConstDataPtr(),sizeof(float)*16,hipMemcpyHostToDevice);

		/*thrust::host_vector<nih::Vector3f> h_points(size*3);
		thrust::host_vector<nih::Vector3f> h_centers(size);
		thrust::host_vector<nih::Bbox3f> h_boxes(size);
		nih::Bbox3f gbox;
		for(int i=0; i<size; i++)
		{
		uint32 tOffset = i*3;
		for( int j=0; j<3; j++)
		{
		for( int k=0; k<3; k++)
		{
		h_points[tOffset+j][k] = vertices[indices[tOffset+j]*3+k];
		h_centers[i][k] += h_points[tOffset+j][k];
		}
		h_boxes[i].insert(h_points[tOffset+j]);
		}			
		}
		for(int i=0; i<h_points.size(); i++)
		std::cout<<h_points[i][0]<<","<<h_points[i][1]<<","<<h_points[i][2]<<std::endl;
		*/
		size_t n_blocks = GridSize(size);
		LoadOBJKernel<<<n_blocks,128>>>(d_vertices,d_indices,d_matrix,d_points,d_centers,d_boxes,size);
		hipFree(d_indices);
		hipFree(d_vertices);
		hipFree(d_matrix);
	}

	//
	size_t BuildBvh(iglu::IGLUOBJReader::Ptr* objs, iglu::IGLUMatrix4x4::Ptr matrixes, size_t objSize)
	{
		using namespace nih;
		uint32 size = 0;
		uint32 *offsets = new uint32[objSize+1];

		for(int i=0; i<objSize; i++)
		{
			offsets[i] = size;
			size += objs[i]->GetTriangleCount();

		}
		thrust::device_vector<Vector3f> d_pointsVec(size*3);
		thrust::host_vector<Bbox3f> h_boxesVec(size);
		thrust::device_vector<Bbox3f> d_boxesVec = h_boxesVec;
		thrust::device_vector<Vector3f> d_centersVec(size,nih::Vector3f(0.f));

		for(int i=0; i<objSize; i++)
		{
			Vector3f* d_points = thrust::raw_pointer_cast(&d_pointsVec[offsets[i]*3]);
			Vector3f* d_centers = thrust::raw_pointer_cast(&d_centersVec[offsets[i]]);
			Bbox3f* d_boxes = thrust::raw_pointer_cast(&d_boxesVec[offsets[i]]);

			LoadOBJReader(objs[i],matrixes+i,d_points,d_centers,d_boxes);
		}
		delete[] offsets;
		/*thrust::host_vector<Vector3f> h_points(d_centersVec);
		for(int i=0; i<h_points.size(); i++)
		std::cout<<h_points[i][0]<<","<<h_points[i][1]<<","<<h_points[i][2]<<std::endl;
		h_points = (d_pointsVec);
		std::cout<<"-------\n";
		for(int i=0; i<h_points.size(); i++)
		std::cout<<h_points[i][0]<<","<<h_points[i][1]<<","<<h_points[i][2]<<std::endl;*/

		//������Χ��
		nih::Bbox3f h_gBox = thrust::reduce(d_boxesVec.begin(),d_boxesVec.end(),nih::Bbox3f(),Add_Bbox<nih::Vector3f>());
		
		/*std::cout<<"�������obj��������"<<std::endl;
		thrust::host_vector<nih::Vector3f> h_pointsVec(d_pointsVec);
		for(int i=0; i<h_pointsVec.size();i+=3)
		{
			std::cout<<i<<std::endl;
			for(int j=0;j<3;j++)
				std::cout<<h_pointsVec[i+j][0]<<","<<h_pointsVec[i+j][1]<<","<<h_pointsVec[i+j][2]<<std::endl;
		}*/


		thrust::device_vector<Bvh_Node> nodes(size-1);
		thrust::device_vector<Bvh_Node> leaves(size);
		hipcub::CachingDeviceAllocator allocator(true);
		KBvh_Builder builder(nodes,leaves,allocator);
		cuda::DBVH h_bvh;
		builder.build(h_gBox,d_centersVec.begin(),d_centersVec.end(),d_boxesVec.begin(),d_boxesVec.end(),&h_bvh);

		thrust::host_vector<Bvh_Node> h_nodes(nodes);
		thrust::host_vector<Bvh_Node> h_leaves(leaves);	
		thrust::host_vector<Bbox3f> h_nodeBoxes(size-1);
		thrust::host_vector<Bbox3f> h_leafBoxes(size);

		//
		//for(int i = 0; i<h_nodes.size(); i++)
		//{ 
		//std::cout<<" parent idx is "<<h_nodes[i].parentIdx<<" ,";

		//if(h_nodes[i].l_isleaf)
		//{
		//std::cout<<i<<" left child "<<" is leaf "<<h_nodes[i].getChild(0);
		//}
		//else
		//{
		//std::cout<<i<<" left child "<<" is internal "<<h_nodes[i].getChild(0);				

		//}
		//if(h_nodes[i].r_isleaf)
		//{
		//std::cout<<" right child "<<" is leaf "<<h_nodes[i].getChild(1)<<std::endl;
		//}
		//else
		//{
		//std::cout<<" right child "<<" is internal "<<h_nodes[i].getChild(1)<<std::endl;
		//}
		//}
		//for(int i=0; i<h_leaves.size(); i++)
		//{
		//std::cout<<i<<" parent is "<<h_leaves[i].parentIdx<<std::endl;
		//std::cout<<" pid is "<<h_leaves[i].pid<<std::endl;
		//}

		h_nodeBoxes.resize(size-1);
		Bbox3f* p_nodeBoxes = thrust::raw_pointer_cast(&h_nodeBoxes.front());
		hipMemcpy(p_nodeBoxes,builder.getNodeBoxes(),sizeof(Bbox3f)*(size-1),hipMemcpyDeviceToHost);

		h_leafBoxes.resize(size);
		Bbox3f* p_leafBoxes = thrust::raw_pointer_cast(&h_leafBoxes.front());
		hipMemcpy(p_leafBoxes,builder.getLeafBoxes(),sizeof(Bbox3f)*(size),hipMemcpyDeviceToHost);
		//ת��ΪDFS���е���
		Bintree_node* nbvh;
		uint32 nbvh_size = size*2-1;
		nbvh = new Bintree_node[nbvh_size];	
		cuda::DFSBintree(&h_nodes,&h_leaves,&h_nodeBoxes,&h_leafBoxes,nbvh);		


		Bintree_node* d_nbvh;
		hipMalloc((void**)&d_nbvh,sizeof(Bintree_node)*nbvh_size);
		hipMemcpy(d_nbvh,nbvh,sizeof(Bintree_node)*nbvh_size,hipMemcpyHostToDevice);
		hipBindTexture( NULL, bvhTex,
			d_nbvh, sizeof(Bintree_node)*nbvh_size );

		hipMalloc((void**)&gd_indices,sizeof(uint32)*size);
		hipMemcpy(gd_indices,builder.getIndices(),sizeof(uint32)*size,hipMemcpyDeviceToDevice);
		hipBindTexture(NULL,indexTex,gd_indices,sizeof(uint32)*size);
		return size;
	}
	size_t VirtualFrustumCulling(size_t triSize,iglu::vec3& eye, float farD, iglu::IGLUOBJReader::Ptr* objs, iglu::IGLUMatrix4x4::Ptr matrixes, size_t objSize,const unsigned int*inElemBuffer, unsigned int * outElemBuffer)
	{
		using namespace nih;
		uint32 size = 0;
		uint32 *offsets = new uint32[objSize+1];
		Vector3f veye(eye.GetConstDataPtr());
		Vector3f* d_eye = NULL;
		hipMalloc((void**)&d_eye,sizeof(Vector3f));
		hipMemcpy(d_eye,&veye,sizeof(Vector3f),hipMemcpyHostToDevice);
		for(int i=0; i<objSize; i++)
		{
			offsets[i] = size;
			size += objs[i]->GetTriangleCount();

		}
		thrust::device_vector<Vector3f> d_pointsVec(size*3);
		thrust::device_vector<TriFrustum> d_frustumVec(size);
		thrust::host_vector<TriFrustum> h_frustumVec(size);
		thrust::host_vector<Vector3f> h_pointsVec(size*3);

		for( int i=0; i<objSize; i++)
		{
			float* matrix = matrixes[i].GetDataPtr();
			std::vector<iglu::vec3> vertices = objs[i]->GetVertecies();
			std::vector<iglu::IGLUOBJTri*> triangles = objs[i]->GetTriangles();
			uint32 offset = offsets[i]*3;
			for(int j=0; j<triangles.size(); j++)
			{			
				for( int k=0; k<3; k++)
				{
					Vector3f p(vertices[triangles[j]->vIdx[k]].GetConstDataPtr());

					h_pointsVec[offset++] = MatrixXVector3f(matrix,p);			
				}
			}
		}
		d_pointsVec = h_pointsVec;
		size_t n_blocks = GridSize(size);
		Vector3f* d_p123 = thrust::raw_pointer_cast(&d_pointsVec.front());
		TriFrustum* d_frustums = thrust::raw_pointer_cast(&d_frustumVec.front());
		/*for(int i=0; i<size; i++)
		{
			GenerateVirFrustum(i,veye,h_pointsVec[i*3],h_pointsVec[i*3+1],h_pointsVec[i*3+2],farD,h_frustumVec[i]);
		}*/
		GenerateVirFrustumKernel<<<n_blocks,128>>>(d_eye,d_p123,d_frustums,farD, size);
		thrust::device_vector<TriFrustum>::iterator values_end = thrust::remove_if(d_frustumVec.begin(),d_frustumVec.end(),is_frustum());
		// since the values after values_end are garbage, we'll resize the vector
		d_frustumVec.resize(values_end - d_frustumVec.begin());
		//std::cout<<"frustum size "<<d_frustumVec.size()<<std::endl;
		/*h_frustumVec = d_frustumVec;
		TriFrustum f = h_frustumVec[0];*/
		//culling 
		size_t frustumSize = d_frustumVec.size();
		thrust::device_vector<cullingContext> d_vectorf(triSize * frustumSize);
		cullingContext* d_list = thrust::raw_pointer_cast(&d_vectorf.front());
		n_blocks = GridSize(frustumSize);
		TriFrustum* d_tfrustumPtr = thrust::raw_pointer_cast(&d_frustumVec.front());
		FrustumCullingKernel<<<n_blocks,128>>>(d_tfrustumPtr,d_frustumVec.size(), triSize,d_list);

		size_t inCount = thrust::count_if(thrust::device,d_vectorf.begin(),d_vectorf.end(),is_valid());		
		thrust::device_vector<cullingContext>fresult(inCount);
		
		thrust::copy_if(d_vectorf.begin(),d_vectorf.end(),fresult.begin(),is_valid());

		//cullingContext* cullingResult = NULL;
		hipMalloc((void**)&cullingResult,sizeof(cullingContext)*inCount);
		hipMemcpy(cullingResult,thrust::raw_pointer_cast(&fresult.front()),sizeof(uint32)*inCount*2,hipMemcpyDeviceToDevice);

	
		hipBindTexture(NULL,virObjTex,cullingResult,sizeof(uint32)*inCount*2);
		//n_blocks = GridSize(fresult.size());
		
	    //std::cout<<fresult.size()<<std::endl;
		/*thrust::host_vector<cullingContext> h_result(fresult);
		for(int i=0; i<h_result.size(); i++)
		{
			std::cout<<h_result[i].frustumId<<":"<<h_result[i].triId<<std::endl;

		}*/

		//UpdateElementKernel<<<n_blocks,128>>>(inElemBuffer,outElemBuffer,thrust::raw_pointer_cast(&fresult.front()),fresult.size());
		hipFree(d_eye);
		//hipFree(gd_indices);
		delete[] offsets;

		return fresult.size();
	}
	void UpdateVirtualObject(unsigned* inPtr, unsigned* outPtr,unsigned size)
	{
		size_t n_blocks = GridSize(size);

		UpdateElementKernel<<<n_blocks,128>>>(inPtr,outPtr,NULL,size);
		hipFree(cullingResult);
	}
	void GenVirtualFrustums(iglu::vec3& eye, float farD, iglu::IGLUOBJReader::Ptr* objs, iglu::IGLUMatrix4x4::Ptr matrixes, size_t objSize)
	{
		using namespace nih;
		uint32 size = 0;
		uint32 *offsets = new uint32[objSize+1];
		Vector3f veye(eye.GetConstDataPtr());
		Vector3f* d_eye = NULL;
		hipMalloc((void**)&d_eye,sizeof(Vector3f));
		hipMemcpy(d_eye,&veye,sizeof(Vector3f),hipMemcpyHostToDevice);
		for(int i=0; i<objSize; i++)
		{
			offsets[i] = size;
			size += objs[i]->GetTriangleCount();

		}
		thrust::device_vector<Vector3f> d_pointsVec(size*3);
		thrust::device_vector<TriFrustum> d_frustumVec(size);
		thrust::host_vector<Vector3f> h_pointsVec(size*3);

		for( int i=0; i<objSize; i++)
		{
			std::vector<iglu::vec3> vertices = objs[i]->GetVertecies();
			std::vector<iglu::IGLUOBJTri*> triangles = objs[i]->GetTriangles();
			uint32 offset = offsets[i]*3;
			for(int j=0; j<triangles.size(); j++)
			{			
				for( int k=0; k<3; k++)
				{
					Vector3f p(vertices[triangles[i]->vIdx[k]].GetConstDataPtr());
					h_pointsVec[offset++] = p;				
				}
			}
		}
		d_pointsVec = h_pointsVec;
		size_t n_blocks = GridSize(size);
		Vector3f* d_p123 = thrust::raw_pointer_cast(&d_pointsVec.front());
		TriFrustum* d_frustums = thrust::raw_pointer_cast(&d_frustumVec.front());
		GenerateVirFrustumKernel<<<n_blocks,128>>>(d_eye,d_p123,d_frustums,farD, size);
		thrust::device_vector<TriFrustum>::iterator values_end = thrust::remove_if(d_frustumVec.begin(),d_frustumVec.end(),is_frustum());
		// since the values after values_end are garbage, we'll resize the vector
		d_frustumVec.resize(values_end - d_frustumVec.begin());

		hipFree(d_eye);
		delete[] offsets;
	}
}