#include "hip/hip_runtime.h"
#include "CudaFunctions.h"
#include <thrust/host_vector.h>
#include "BVH\cuda_klbvh.h"

#include <iostream>
texture<uint32> bvhTex;
namespace cuda
{
	
	//@vertices �Ƕ������飬ÿ��������4����������ʾ
	//@indices �������ζ�����������飬ÿ����������3������
	//@matrix ��4��4ģ�;���
	//@points ������������������ζ���
	//@centers ������������������ε�����
	//@boxes ������������������εİ�Χ��
	//@size �����θ���
	__global__ void LoadOBJKernel(float* vertices,
		uint32* indices,
		float* matrix,
		nih::Vector3f* points,
		nih::Vector3f* centers,
		nih::Bbox3f* boxes,
		uint32 size)
	{
		uint32 step = blockDim.x * gridDim.x;
		for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
			i < size; 
			i += step) 
		{
			uint32 tOffset = 3*i;


			for( int j=0; j<3; j++)
			{
				for( int k=0; k<3; k++)
				{
					points[tOffset+j][k] = vertices[indices[tOffset+j]*3+k];
					centers[i][k] += points[tOffset+j][k];
				}
				boxes[i].insert(points[tOffset+j]);				
			}			
			centers[i] /= 3.f;
		}
	}


	void LoadOBJReader(iglu::IGLUOBJReader::Ptr obj, 
		iglu::IGLUMatrix4x4::Ptr matrix,
		nih::Vector3f* d_points,
		nih::Vector3f* d_centers,
		nih::Bbox3f* d_boxes)
	{
		size_t size = obj->GetTriangleCount();
		size_t vertSize = obj->GetVaoVerts().size();
		float* vertices = (float*)(&obj->GetVaoVerts()[0]);
		uint32* indices = obj->GetElementArrayData();
		for(int i=0; i<size*3; i+=3)
			std::cout<<indices[i]<<","<<indices[i+1]<<","<<indices[i+2]<<std::endl;
		float* d_vertices;
		hipMalloc((void**)&d_vertices,sizeof(float)*12*size);
		hipMemcpy(d_vertices,vertices,sizeof(float)*12*size,hipMemcpyHostToDevice);

		uint32* d_indices;
		hipMalloc((void**)&d_indices,sizeof(uint32)*3*size);
		hipMemcpy(d_indices,indices,sizeof(uint32)*3*size,hipMemcpyHostToDevice);

		/*thrust::host_vector<nih::Vector3f> h_points(size*3);
		thrust::host_vector<nih::Vector3f> h_centers(size);
		thrust::host_vector<nih::Bbox3f> h_boxes(size);
		nih::Bbox3f gbox;
		for(int i=0; i<size; i++)
		{
		uint32 tOffset = i*3;
		for( int j=0; j<3; j++)
		{
		for( int k=0; k<3; k++)
		{
		h_points[tOffset+j][k] = vertices[indices[tOffset+j]*3+k];
		h_centers[i][k] += h_points[tOffset+j][k];
		}
		h_boxes[i].insert(h_points[tOffset+j]);
		}			
		}
		for(int i=0; i<h_points.size(); i++)
		std::cout<<h_points[i][0]<<","<<h_points[i][1]<<","<<h_points[i][2]<<std::endl;
		*/
		size_t blockSize = 128;
		int numSMs;
		hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);
		size_t max_blocks = 65535;
		size_t n_blocks   = nih::min( max_blocks, (size + (blockSize*numSMs)-1) / (blockSize*numSMs) );



		LoadOBJKernel<<<n_blocks,blockSize>>>(d_vertices,d_indices,&matrix[0][0],d_points,d_centers,d_boxes,size);
		hipFree(d_indices);
		hipFree(d_vertices);
	}
	
	//
	void BuildBvh(iglu::IGLUOBJReader::Ptr* objs, iglu::IGLUMatrix4x4::Ptr matrixes, size_t objSize)
	{
		using namespace nih;
		uint32 size = 0;
		uint32 *offsets = new uint32[objSize+1];

		for(int i=0; i<objSize; i++)
		{
			offsets[i] = size;
			size += objs[i]->GetTriangleCount();

		}
		thrust::device_vector<Vector3f> d_pointsVec(size*3);
		thrust::host_vector<Bbox3f> h_boxesVec(size);
		thrust::device_vector<Bbox3f> d_boxesVec = h_boxesVec;
		thrust::device_vector<Vector3f> d_centersVec(size,nih::Vector3f(0.f));

		for(int i=0; i<objSize; i++)
		{
			Vector3f* d_points = thrust::raw_pointer_cast(&d_pointsVec[offsets[i]*3]);
			Vector3f* d_centers = thrust::raw_pointer_cast(&d_centersVec[offsets[i]]);
			Bbox3f* d_boxes = thrust::raw_pointer_cast(&d_boxesVec[offsets[i]]);

			LoadOBJReader(objs[i],matrixes,d_points,d_centers,d_boxes);
		}
		delete[] offsets;
		/*thrust::host_vector<Vector3f> h_points(d_centersVec);
		for(int i=0; i<h_points.size(); i++)
		std::cout<<h_points[i][0]<<","<<h_points[i][1]<<","<<h_points[i][2]<<std::endl;
		h_points = (d_pointsVec);
		std::cout<<"-------\n";
		for(int i=0; i<h_points.size(); i++)
		std::cout<<h_points[i][0]<<","<<h_points[i][1]<<","<<h_points[i][2]<<std::endl;*/

		//������Χ��
		nih::Bbox3f h_gBox = thrust::reduce(d_boxesVec.begin(),d_boxesVec.end(),nih::Bbox3f(),Add_Bbox<nih::Vector3f>());

		thrust::device_vector<Bvh_Node> nodes(size-1);
		thrust::device_vector<Bvh_Node> leaves(size);
		hipcub::CachingDeviceAllocator allocator(true);
		KBvh_Builder builder(nodes,leaves,allocator);
		cuda::DBVH h_bvh;
		builder.build(h_gBox,d_centersVec.begin(),d_centersVec.end(),d_boxesVec.begin(),d_boxesVec.end(),&h_bvh);

		thrust::host_vector<Bvh_Node> h_nodes(nodes);
		thrust::host_vector<Bvh_Node> h_leaves(leaves);	
		thrust::host_vector<Bbox3f> h_nodeBoxes(size-1);
		thrust::host_vector<Bbox3f> h_leafBoxes(size);

		/*
		for(int i = 0; i<h_nodes.size(); i++)
		{ 
		std::cout<<" parent idx is "<<h_nodes[i].parentIdx<<" ,";

		if(h_nodes[i].l_isleaf)
		{
		std::cout<<i<<" left child "<<" is leaf "<<h_nodes[i].getChild(0);
		}
		else
		{
		std::cout<<i<<" left child "<<" is internal "<<h_nodes[i].getChild(0);				

		}
		if(h_nodes[i].r_isleaf)
		{
		std::cout<<" right child "<<" is leaf "<<h_nodes[i].getChild(1)<<std::endl;
		}
		else
		{
		std::cout<<" right child "<<" is internal "<<h_nodes[i].getChild(1)<<std::endl;
		}
		}
		for(int i=0; i<h_leaves.size(); i++)
		{
		std::cout<<i<<" parent is "<<h_leaves[i].parentIdx<<std::endl;
		std::cout<<" pid is "<<h_leaves[i].pid<<std::endl;
		}*/





		h_nodeBoxes.resize(size-1);
		Bbox3f* p_nodeBoxes = thrust::raw_pointer_cast(&h_nodeBoxes.front());
		hipMemcpy(p_nodeBoxes,builder.getNodeBoxes(),sizeof(Bbox3f)*(size-1),hipMemcpyDeviceToHost);

		h_leafBoxes.resize(size);
		Bbox3f* p_leafBoxes = thrust::raw_pointer_cast(&h_leafBoxes.front());
		hipMemcpy(p_leafBoxes,builder.getLeafBoxes(),sizeof(Bbox3f)*(size),hipMemcpyDeviceToHost);
		//ת��ΪDFS���е���
		Bintree_node* nbvh;
		uint32 nbvh_size = size*2-1;
		nbvh = new Bintree_node[nbvh_size];	
		cuda::DFSBintree(&h_nodes,&h_leaves,&h_nodeBoxes,&h_leafBoxes,nbvh);		


		Bintree_node* d_nbvh;
		hipMalloc((void**)&d_nbvh,sizeof(Bintree_node)*nbvh_size);
		hipMemcpy(d_nbvh,nbvh,sizeof(Bintree_node)*nbvh_size,hipMemcpyHostToDevice);
		hipBindTexture( NULL, bvhTex,
			d_nbvh, sizeof(Bintree_node)*nbvh_size );
	}

	void GenVirtualFrustums(iglu::IGLUOBJReader::Ptr* objs, iglu::IGLUMatrix4x4::Ptr matrixes, size_t objSize)
	{
		using namespace nih;
		uint32 size = 0;
		uint32 *offsets = new uint32[objSize+1];

		for(int i=0; i<objSize; i++)
		{
			offsets[i] = size;
			size += objs[i]->GetTriangleCount();

		}
		thrust::device_vector<Vector3f> d_pointsVec(size*3);
		thrust::device_vector<TriFrustum> d_frustums(size);
		delete[] offsets;
	}
}