#include "VORApp.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

iglu::IGLUApp* app;
int main()
{
	using namespace OGL;
   
	app = new VORApp("../../CommonSampleFiles/scenes/virObjRef.txt");
	app->Run();
    return 0;
}


